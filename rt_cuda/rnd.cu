#include "hip/hip_runtime.h"
#include "rnd.h"
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#include <>
#include <hiprand/hiprand_kernel.h>


__global__ void rnd_init(size_t _count, hiprandState * _state)
{
	int index = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y + (blockIdx.x * blockDim.x * blockDim.y * blockDim.z);
	if (index >= _count)
	{
		return;
	}
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, index, 0, &_state[index]);
}

__host__ Rnd::Rnd(const Rnd & other) : _owner(false)
{
	_state = other._state;
	_count = other._count;	
}

__host__ Rnd::Rnd(dim3 blocks, dim3 threads) : _state(nullptr), _owner(true)
{		
	_count = blocks.x * blocks.y * threads.x * threads.y * threads.z;
	hipMalloc(&_state, _count * sizeof(hiprandState));
	rnd_init <<<blocks, threads>>> (_count, _state);
}

__host__ Rnd::~Rnd()
{
	if (_owner)
	{
		hipFree(&_state);
	}
}

__device__ float Rnd::random()
{
	int index = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y + (blockIdx.x * blockDim.x * blockDim.y * blockDim.z);
	if (index >= _count)
	{
		return 0;
	}
	return hiprand_uniform(&_state[index]);
}

__device__ vec3 Rnd::random_in_unit_disk()
{
	vec3 p;
	do {
		p = 2.0*vec3(random(), random(), 0) - vec3(1, 1, 0);
	} while (dot(p, p) >= 1.0);
	return p;
}

__device__ vec3 Rnd::random_in_unit_sphere()
{
	vec3 p;
	do {
		p = 2.0*vec3(random(), random(), random()) - vec3(1, 1, 1);
	} while (p.squared_length() >= 1.0);
	return p;
}


